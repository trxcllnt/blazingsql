#include "GPUManager.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudf/utilities/error.hpp>

namespace ral {
namespace config {

size_t gpuFreeMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	// CUDF_CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}

size_t gpuTotalMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	// CUDF_CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return total;
}

size_t gpuUsedMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	// CUDF_CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return total - free;
}

}  // namespace config
}  // namespace ral
